#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}

//GPU kernel SAXPY operation
__global__
void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}

//CPU SAXPY Operation
void saxpy_host(int n, float a, float *x, float *y) {
    for (int i = 0; i < n; ++i)
        y[i] = a*x[i] + y[i];
}

int main(void) {
    // Num of elements
    int N = 1000;
    //Block size and number
    int block_size, block_no;
    //Vectors operators
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    block_size = 256; //threads per block
    block_no = (N)/block_size; //Numm of blocks

    //Work definition
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(block_no, 1, 1);

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //Escalar operator
    float a = 2.0f;

    printf("Allocating device memory on host..\n");
    //GPU memory allocation
    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    printf("Copying to device..\n");
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    //SAXPY on CUDA
    clock_t start_d=clock();

    printf("Doing GPU saxpy\n");
    saxpy<<<block_no,block_size>>>(N, a, d_x, d_y);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    clock_t end_d = clock();

    printf("Doing CPU saxpy\n");
    clock_t start_h = clock();
    saxpy_host(N, a, x, y);
    clock_t end_h = clock();

    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    printf("n = %d \t GPU time = %fs \t CPU time = %fs\n", N, time_d, tim_h);

    //Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    return 0;
}